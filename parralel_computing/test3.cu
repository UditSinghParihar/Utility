#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 5

__global__ void add(int *a, int *b, int *c) {
	*c += a[threadIdx.x] * b[threadIdx.x];
}

void print_five(int* array){
	for(int i=0; i<5; ++i){
		printf("%d ", array[i]);
	}
	printf("\n");
}

void random_ints(int *a, int n){
   int i;
   for (i = 0; i < n; ++i)
       a[i] = rand() %10;
}

int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	*c = 0;
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, sizeof(int), hipMemcpyHostToDevice);
	
	// Launch add() kernel on GPU with N blocks
	add<<<1,N>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	print_five(a);
	print_five(b);
	pritf("c: %d\n", c);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b);
	return 0;
}