#include <stdio.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

void print_few(int* array){
	for(int i=0; i<5; ++i){
		printf("%d ", array[i]);
	}
	printf("\n");
}

void random_ints(int *a, int n){
   int i;
   for (i = 0; i < n; ++i)
       a[i] = rand() %10;
}

int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	print_few(a);
	print_few(b);
	print_few(c);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}